#include "WireCellCuda/cuFftDFT.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <assert.h>
#include <iostream>

using namespace WireCell;
using complex_t = IDFT::complex_t;

/* Because otherwise a more explicit casting will be needed which
 * will reduce performance. */
static_assert(sizeof(hipfftComplex) == sizeof(complex_t));

Cuda::memArgs<hipfftComplex>
Cuda::cuFftDFT::preTransformTasks(
    Cuda::memArgs<complex_t> hmem, int size
) const
{
    // Allocate memory for the input and output arrays in the GPU   
    static Cuda::memArgs<hipfftComplex> dmem;
    assert(hipMalloc( &dmem.in, size*sizeof(hipfftComplex) ) == hipSuccess);
    if( dmem.in == dmem.out ) dmem.out = const_cast<hipfftComplex*>(dmem.in);
    else assert(hipMalloc( &dmem.out, size*sizeof(hipfftComplex) ) == hipSuccess);

    // Copy input array in CPU to the GPU
    assert(hipMemcpy(
        (void *)dmem.in, (void *)hmem.in, 
        size*sizeof(hipfftComplex), 
        hipMemcpyHostToDevice
    ) == hipSuccess);

    return dmem;
}

void Cuda::cuFftDFT::postTransformTasks(
    memArgs<complex_t> hmem, memArgs<hipfftComplex> dmem, int size
) const
{
    // Copy result back to the CPU
    assert(hipMemcpy(
        (void *)hmem.out, (void *)dmem.out, 
        size*sizeof(complex_t), 
        hipMemcpyDeviceToHost
    ) == hipSuccess);

    // Free memory on GPU
    assert(hipFree((void *)dmem.in) == hipSuccess);
    if( dmem.in != dmem.out )
        assert(hipFree((void *)dmem.out) == hipSuccess);
}

void Cuda::cuFftDFT::gen1d(const complex_t* in, complex_t* out, int size, int dir) const
{
    static hipfftHandle plan;
    Cuda::memArgs<complex_t> hmem{in, out};
    Cuda::memArgs<hipfftComplex> dmem = preTransformTasks(hmem, size);

    // Create the plan and perform the transform
    assert(hipfftPlan1d(&plan, size, HIPFFT_C2C, 1) == HIPFFT_SUCCESS);
    assert(hipfftExecC2C(plan, const_cast<hipfftComplex*>(dmem.in), dmem.out, dir) == HIPFFT_SUCCESS);
    
    postTransformTasks(hmem, dmem, size);
}

void Cuda::cuFftDFT::fwd1d(const complex_t* in, complex_t* out, int size) const
{ gen1d(in, out, size, HIPFFT_FORWARD); }

void Cuda::cuFftDFT::inv1d(const complex_t* in, complex_t* out, int size) const 
{ gen1d(in, out, size, HIPFFT_BACKWARD); }

void Cuda::cuFftDFT::fwd1b(const complex_t* in, complex_t* out, int nrows, int ncols, int axis) const {}
void Cuda::cuFftDFT::inv1b(const complex_t* in, complex_t* out, int nrows, int ncols, int axis) const {}

void Cuda::cuFftDFT::fwd2d(const complex_t* in, complex_t* out, int nrows, int ncols) const {}
void Cuda::cuFftDFT::inv2d(const complex_t* in, complex_t* out, int nrows, int ncols) const {}

void Cuda::cuFftDFT::transpose(const scalar_t* in, scalar_t* out, int nrows, int ncols) const {}
void Cuda::cuFftDFT::transpose(const complex_t* in, complex_t* out, int nrows, int ncols) const {}