#include "WireCellCuda/cuFftDFT.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <assert.h>
#include <iostream>

using namespace WireCell;
using complex_t = IDFT::complex_t;

/* Because otherwise a more explicit casting will be needed which
 * will reduce performance. */
static_assert(sizeof(hipfftComplex) == sizeof(complex_t));

Cuda::memArgs<hipfftComplex>
Cuda::cuFftDFT::preTransformTasks(
    Cuda::memArgs<complex_t> hmem, int size
) const
{
    // Allocate memory for the input and output arrays in the GPU   
    static Cuda::memArgs<hipfftComplex> dmem;

    assert(hipMalloc( &dmem.in, size*sizeof(hipfftComplex) ) == hipSuccess);

    if( dmem.in == dmem.out ) dmem.out = const_cast<hipfftComplex*>(dmem.in);
    else assert(hipMalloc( &dmem.out, size*sizeof(hipfftComplex) ) == hipSuccess);

    // Copy input array in CPU to the GPU
    assert(hipMemcpy(
        (void *)dmem.in, (void *)hmem.in, 
        size*sizeof(hipfftComplex), 
        hipMemcpyHostToDevice
    ) == hipSuccess);

    return dmem;
}

void Cuda::cuFftDFT::postTransformTasks(
    memArgs<complex_t> hmem, memArgs<hipfftComplex> dmem, int size
) const
{
    // Copy result back to the CPU
    assert(hipMemcpy(
        (void *)hmem.out, (void *)dmem.out, 
        size*sizeof(complex_t), 
        hipMemcpyDeviceToHost
    ) == hipSuccess);

    // Free memory on GPU
    assert(hipFree((void *)dmem.in) == hipSuccess);
    if( dmem.in != dmem.out )
        assert(hipFree((void *)dmem.out) == hipSuccess);
}

void Cuda::cuFftDFT::gen1d(const complex_t* in, complex_t* out, int size, int dir) const
{
    static hipfftHandle plan;
    Cuda::memArgs<complex_t> hmem{in, out};
    Cuda::memArgs<hipfftComplex> dmem = preTransformTasks(hmem, size);

    // Create the plan and perform the transform
    assert(hipfftPlan1d(&plan, size, HIPFFT_C2C, 1) == HIPFFT_SUCCESS);
    assert(hipfftExecC2C(plan, const_cast<hipfftComplex*>(dmem.in), dmem.out, dir) == HIPFFT_SUCCESS);
    
    postTransformTasks(hmem, dmem, size);
}

void Cuda::cuFftDFT::gen1b(const complex_t* in, complex_t* out, int nrows, int ncols, int axis, int dir) const
{
    static hipfftHandle plan;
    Cuda::memArgs<complex_t> hmem{in, out};
    Cuda::memArgs<hipfftComplex> dmem = preTransformTasks(hmem, nrows*ncols);

    // Create the plan and perform the transform
    assert(axis == 0 || axis == 1);
    if( axis == 0 )
    {
        assert(hipfftPlanMany(
            &plan, 1, new int[2] {ncols, nrows}, 
            &nrows, 1, ncols, 
            &nrows, 1, ncols, 
            HIPFFT_C2C, nrows 
        ) == HIPFFT_SUCCESS);
    } else
    {
        assert(hipfftPlanMany(
            &plan, 1, new int[2] {ncols, nrows}, 
            &ncols, nrows, 1, 
            &ncols, nrows, 1, 
            HIPFFT_C2C, nrows 
        ) == HIPFFT_SUCCESS);
    }

    assert(hipfftExecC2C(plan, const_cast<hipfftComplex*>(dmem.in), dmem.out, dir) == HIPFFT_SUCCESS);
    
    postTransformTasks(hmem, dmem, nrows*ncols);
}

void Cuda::cuFftDFT::gen2d(const complex_t* in, complex_t* out, int nrows, int ncols, int dir) const
{
    static hipfftHandle plan;
    Cuda::memArgs<complex_t> hmem{in, out};
    Cuda::memArgs<hipfftComplex> dmem = preTransformTasks(hmem, nrows*ncols);

    assert(hipfftPlan2d(&plan, nrows, ncols, HIPFFT_C2C) == HIPFFT_SUCCESS);
    assert(hipfftExecC2C(plan, const_cast<hipfftComplex*>(dmem.in), dmem.out, dir) == HIPFFT_SUCCESS);
    
    postTransformTasks(hmem, dmem, nrows*ncols);
}

void Cuda::cuFftDFT::fwd1d(const complex_t* in, complex_t* out, int size) const
{ gen1d(in, out, size, HIPFFT_FORWARD); }

void Cuda::cuFftDFT::inv1d(const complex_t* in, complex_t* out, int size) const 
{ gen1d(in, out, size, HIPFFT_BACKWARD); }

void Cuda::cuFftDFT::fwd1b(const complex_t* in, complex_t* out, int nrows, int ncols, int axis) const 
{ gen1b(in, out, nrows, ncols, axis, HIPFFT_FORWARD); }

void Cuda::cuFftDFT::inv1b(const complex_t* in, complex_t* out, int nrows, int ncols, int axis) const 
{ gen1b(in, out, nrows, ncols, axis, HIPFFT_BACKWARD); }

void Cuda::cuFftDFT::fwd2d(const complex_t* in, complex_t* out, int nrows, int ncols) const 
{ gen2d(in, out, nrows, ncols, HIPFFT_FORWARD); }

void Cuda::cuFftDFT::inv2d(const complex_t* in, complex_t* out, int nrows, int ncols) const 
{ gen2d(in, out, nrows, ncols, HIPFFT_BACKWARD); }

void Cuda::cuFftDFT::transpose(const scalar_t* in, scalar_t* out, int nrows, int ncols) const {}
void Cuda::cuFftDFT::transpose(const complex_t* in, complex_t* out, int nrows, int ncols) const {}